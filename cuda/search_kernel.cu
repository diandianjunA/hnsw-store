#include "hip/hip_runtime.h"
#include <thrust/binary_search.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>

#include "priority_queue.cuh"
#include "search_kernel.cuh"
#include <time.h>

#define CHECK(res)                                                             \
  {                                                                            \
    if (res != hipSuccess) {                                                  \
      printf("Error ：%s:%d , ", __FILE__, __LINE__);                          \
      printf("code : %d , reason : %s \n", res, hipGetErrorString(res));      \
      exit(-1);                                                                \
    }                                                                          \
  }

__inline__ __device__ unsigned int *get_linklist0(unsigned int internal_id) {
  return (unsigned int *)(data + internal_id * size_data_per_element);
}

__inline__ __device__ unsigned short int getListCount(unsigned int *ptr) {
  return *((unsigned short int *)ptr);
}

__global__ void search_kernel(const float *query_data, int num_query, int k,
                              const int *entry_node, Node *device_pq,
                              bool *visited_table, int *global_candidate_nodes,
                              float *global_candidate_distances, int *found_cnt,
                              int *nns, float *distances) {

  static __shared__ int size;

  // int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

  Node *ef_search_pq = device_pq + ef_search * blockIdx.x;
  int *candidate_nodes = global_candidate_nodes + ef_search * blockIdx.x;
  float *candidate_distances =
      global_candidate_distances + ef_search * blockIdx.x;

  bool *_visited_table = visited_table + num_data * blockIdx.x;

  for (int i = blockIdx.x; i < num_query; i += gridDim.x) {
    if (threadIdx.x == 0) {
      size = 0;
    }
    __syncthreads();

    const float *src_vec = query_data + i * dims;
    PushNodeToSearchPq(ef_search_pq, &size, query_data, entry_node[i]);

    if (CheckVisited(_visited_table, entry_node[i])) {
      continue;
    }
    __syncthreads();

    int idx = GetCand(ef_search_pq, size);
    while (idx >= 0) {
      __syncthreads();
      if (threadIdx.x == 0)
        ef_search_pq[idx].checked = true;
      int entry = ef_search_pq[idx].nodeid;
      __syncthreads();

      unsigned int *entry_neighbor_ptr = get_linklist0(entry);
      int deg = getListCount(entry_neighbor_ptr);

      for (int j = 1; j <= deg; ++j) {
        int dstid = *(entry_neighbor_ptr + j);

        if (CheckVisited(_visited_table, dstid)) {
          continue;
        }
        __syncthreads();

        PushNodeToSearchPq(ef_search_pq, &size, src_vec, dstid);
      }
      __syncthreads();
      idx = GetCand(ef_search_pq, size);
    }
    __syncthreads();

    for (int j = threadIdx.x; j < num_data; j += blockDim.x) {
      _visited_table[j] = false;
    }
    __syncthreads();
    // get sorted neighbors
    if (threadIdx.x == 0) {
      int size2 = size;
      while (size > 0) {
        candidate_nodes[size - 1] = ef_search_pq[0].nodeid;
        candidate_distances[size - 1] = ef_search_pq[0].distance;
        PqPop(ef_search_pq, &size);
      }
      found_cnt[i] = size2 < k ? size2 : k;
      for (int j = 0; j < found_cnt[i]; ++j) {
        nns[j + i * k] = candidate_nodes[j];
        distances[j + i * k] = out_scalar(candidate_distances[j]);
      }
    }
    __syncthreads();
  }
}

__global__ void kernel_check() {
  printf("Hello from kernel\n");

  for (int i = 0; i < num_data; i++) {
    float *data = getDataByInternalId(i);
    printf("data[%d] = [", i);
    for (int j = 0; j < dims; j++) {
      printf("%f, ", data[j]);
    }
    printf("]\n");
  }

  for (int i = 0; i < num_data; i++) {
    unsigned int *linklist = get_linklist0(i);
    int deg = getListCount(linklist);
    printf("linklist[%d] = [", i);
    for (int j = 1; j <= deg; j++) {
      printf("%d, ", *(linklist + j));
    }
    printf("]\n");
  }
}

void cuda_search(int entry_node, const float *query_data, int num_query,
                 int ef_search_, int k, int *nns, float *distances,
                 int *found_cnt) {
  int block_cnt_ = 1024;
  hipMemcpyToSymbol(HIP_SYMBOL(ef_search), &ef_search_, sizeof(int));
  thrust::device_vector<Node> device_pq(ef_search_ * block_cnt_);
  thrust::device_vector<int> global_candidate_nodes(ef_search_ * block_cnt_);
  thrust::device_vector<float> global_candidate_distances(ef_search_ *
                                                          block_cnt_);
  int num_data_ = 0;
  hipMemcpyFromSymbol(&num_data_, HIP_SYMBOL(num_data), sizeof(int));
  thrust::device_vector<bool> device_visited_table(num_data_ * block_cnt_,
                                                   false);
  thrust::device_vector<int> device_found_cnt(num_query);
  thrust::device_vector<int> device_nns(k * num_query);
  thrust::device_vector<float> device_distances(k * num_query);

  int dims_;
  hipMemcpyFromSymbol(&dims_, HIP_SYMBOL(dims), sizeof(int));
  thrust::device_vector<float> device_qdata_(num_query * dims_);
  thrust::copy(query_data, query_data + num_query * dims_,
               device_qdata_.begin());
  thrust::device_vector<int> dev_entries(num_query, entry_node);

  search_kernel<<<block_cnt_, dims_>>>(
      thrust::raw_pointer_cast(device_qdata_.data()), num_query, k,
      thrust::raw_pointer_cast(dev_entries.data()),
      thrust::raw_pointer_cast(device_pq.data()),
      thrust::raw_pointer_cast(device_visited_table.data()),
      thrust::raw_pointer_cast(global_candidate_nodes.data()),
      thrust::raw_pointer_cast(global_candidate_distances.data()),
      thrust::raw_pointer_cast(device_found_cnt.data()),
      thrust::raw_pointer_cast(device_nns.data()),
      thrust::raw_pointer_cast(device_distances.data()));
  CHECK(hipDeviceSynchronize());
  thrust::copy(device_nns.begin(), device_nns.end(), nns);
  thrust::copy(device_distances.begin(), device_distances.end(), distances);
  thrust::copy(device_found_cnt.begin(), device_found_cnt.end(), found_cnt);
  CHECK(hipDeviceSynchronize());
}

__global__ void cuda_init() {
  printf("Hello from kernel\n");
}

void cuda_init(int dims_, char *data_, size_t size_data_per_element_,
               size_t offsetData_, int max_m_, int ef_search_, int num_data_,
               size_t data_size_) {
  hipMemcpyToSymbol(HIP_SYMBOL(dims), &dims_, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(size_data_per_element), &size_data_per_element_,
                     sizeof(size_t));
  hipMemcpyToSymbol(HIP_SYMBOL(offsetData), &offsetData_, sizeof(size_t));
  hipMemcpyToSymbol(HIP_SYMBOL(ef_search), &ef_search_, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(num_data), &num_data_, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(data_size), &data_size_, sizeof(size_t));

  hipMemcpyToSymbol(HIP_SYMBOL(max_m), &max_m_, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(data), &data_, sizeof(char *));
  CHECK(hipDeviceSynchronize());

  // kernel_check<<<1, 1>>>();
  // CHECK(hipDeviceSynchronize());
  // kernel_check2<<<1, 1>>>();
  // CHECK(hipDeviceSynchronize());
}
